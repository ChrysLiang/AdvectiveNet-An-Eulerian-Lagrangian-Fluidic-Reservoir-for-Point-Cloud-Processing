#include "hip/hip_runtime.h"
#include <torch/script.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


//// blockIdx.x: num_points
//// blockIdx.y: num_channel
//// threadIdx.x: batch_size

__global__ void cal_feature_sum_kernel(
    const torch::PackedTensorAccessor<float,3,torch::RestrictPtrTraits,size_t> pc, //// (batch, 3, num_points)
    const torch::PackedTensorAccessor<float,3,torch::RestrictPtrTraits,size_t> feature, //// (batch, num_channel, num_points)
    const int grid_size,
    torch::PackedTensorAccessor<float,5,torch::RestrictPtrTraits,size_t> sum_feature,  //// (batch, num_channel, grid_size, grid_size, grid_size)
    torch::PackedTensorAccessor<int32_t,4,torch::RestrictPtrTraits,size_t> count_pc   //// (batch, grid_size, grid_size, grid_size)
    )
{
  float dx=1.0/grid_size;
  float x = pc[threadIdx.x][0][blockIdx.x];
  float y = pc[threadIdx.x][1][blockIdx.x];
  float z = pc[threadIdx.x][2][blockIdx.x];
  int cell_x = __float2int_rd(x/dx);
  int cell_y = __float2int_rd(y/dx);
  int cell_z = __float2int_rd(z/dx);
  cell_x = min(max(cell_x, 0), grid_size-1);
  cell_y = min(max(cell_y, 0), grid_size-1);
  cell_z = min(max(cell_z, 0), grid_size-1);

  atomicAdd(&(sum_feature[threadIdx.x][blockIdx.y][cell_x][cell_y][cell_z]), feature[threadIdx.x][blockIdx.y][blockIdx.x]);
  atomicAdd(&(count_pc[threadIdx.x][cell_x][cell_y][cell_z]), 1);
}

__global__ void cal_feature_relative_kernel(
    const torch::PackedTensorAccessor<float,3,torch::RestrictPtrTraits,size_t> pc, //// (batch, 3, num_points)
    const torch::PackedTensorAccessor<float,3,torch::RestrictPtrTraits,size_t> feature, //// (batch, num_channel, num_points)
    const torch::PackedTensorAccessor<float,5,torch::RestrictPtrTraits,size_t> sum_feature, //// (batch, 3, grid_size, grid_size, grid_size)
    const torch::PackedTensorAccessor<int32_t,4,torch::RestrictPtrTraits,size_t> count_pc, //// (batch, 3, grid_size, grid_size, grid_size)
    const int grid_size,
    torch::PackedTensorAccessor<float,3,torch::RestrictPtrTraits,size_t> relative_feature   //// (batch, 2*num_channel, num_points)
    )
{
  float dx=1.0/grid_size;
  float x = pc[threadIdx.x][0][blockIdx.x];
  float y = pc[threadIdx.x][1][blockIdx.x];
  float z = pc[threadIdx.x][2][blockIdx.x];
  int cell_x = __float2int_rd(x/dx);
  int cell_y = __float2int_rd(y/dx);
  int cell_z = __float2int_rd(z/dx);
  cell_x = min(max(cell_x, 0), grid_size-1);
  cell_y = min(max(cell_y, 0), grid_size-1);
  cell_z = min(max(cell_z, 0), grid_size-1);

  int count = max(count_pc[threadIdx.x][cell_x][cell_y][cell_z], 1);

  relative_feature[threadIdx.x][blockIdx.y][blockIdx.x]=sum_feature[threadIdx.x][blockIdx.y][cell_x][cell_y][cell_z]/count;
  relative_feature[threadIdx.x][blockIdx.y+blockDim.y][blockIdx.x]=feature[threadIdx.x][blockIdx.y][blockIdx.x]-relative_feature[threadIdx.x][blockIdx.y][blockIdx.x];
}


torch::Tensor cal_feature_relative(torch::Tensor pc, torch::Tensor feature, int grid_size)
{
int batch_size = pc.size(0);
int num_points = pc.size(2);
int num_channel = feature.size(1);
auto sum_feature = torch::zeros({batch_size, num_channel, grid_size, grid_size, grid_size}).to(pc);
auto count_pc = torch::zeros({batch_size, grid_size, grid_size, grid_size}).to(pc).to(at::kInt);
auto relative_feature = torch::zeros({batch_size, num_channel*2, num_points}).to(pc);
pc = (pc + 1) / 2;

const int threads = batch_size;
const dim3 blocks(num_points, num_channel);

cal_feature_sum_kernel<<<blocks, threads>>>(
pc.packed_accessor<float,3,torch::RestrictPtrTraits,size_t>(),
feature.packed_accessor<float,3,torch::RestrictPtrTraits,size_t>(),
grid_size,
sum_feature.packed_accessor<float,5,torch::RestrictPtrTraits,size_t>(),
count_pc.packed_accessor<int32_t,4,torch::RestrictPtrTraits,size_t>()
);

cal_feature_relative_kernel<<<blocks, threads>>>(
pc.packed_accessor<float,3,torch::RestrictPtrTraits,size_t>(),
feature.packed_accessor<float,3,torch::RestrictPtrTraits,size_t>(),
sum_feature.packed_accessor<float,5,torch::RestrictPtrTraits,size_t>(),
count_pc.packed_accessor<int32_t,4,torch::RestrictPtrTraits,size_t>(),
grid_size,
relative_feature.packed_accessor<float,3,torch::RestrictPtrTraits,size_t>()
);


return relative_feature;
}